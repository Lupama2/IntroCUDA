/*
Calculo de forma serial el producto de matrices en base a la tarea siguiente

2) Para aprovechar al máximo la próxima clase, escriban un programa serial en C, que aloque memoria, inicialice y multiplique dos matrices cuadradas de NxN, A y B, de forma que 

C = A.B 

sino se acuerdan googleenlo, o preguntenle a chatgpt :-p. Agreguenle los timers de cpu que usamos en suma vectores, compilen, y si pueden miren como escalea tcpu vs N. ¿Como debería?

Piensen una estrategia para paralelizar la multiplicacion.

*/
//Importo librerías

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <time.h>


#define SIZE 1024


int main(int argc, char **argv)
{
	int N;
    //Para permitir que el usuario ingrese el tamaño
	if(argc==2) N=atoi(argv[1]);
	else N=SIZE;
    // printf("N = {%d}", N);

    //Creo punteos a las matrices de tamaño NxN
    int **A, **B, **C;

    //Aloco memoria
    A = (int**) malloc(N * sizeof(int*));
    for(int i = 0; i < N; i++){
        A[i] = (int*) malloc(N * sizeof(int));
    }

    B = (int**) malloc(N * sizeof(int*));
    for(int i = 0; i < N; i++){
        B[i] = (int*) malloc(N * sizeof(int));
    }

    C = (int**) malloc(N * sizeof(int*));
    for(int i = 0; i < N; i++){
        C[i] = (int*) malloc(N * sizeof(int));
    }

    //Asigno nros aleatorios a las matrices A y B
    for(int i = 0;i < N; i += 1){
        for(int j = 0; j < N; j += 1){
            A[i][j] = rand()%10;
            B[i][j] = rand()%10;
        }
    }


    //Multiplico las matrices A*B

    //Calculo el tiempo de cómputo
    clock_t start = clock();

    

    //Accedo a cada elemento de C
    for(int i = 0;i < N; i += 1){
        for(int j = 0; j < N; j += 1){
            //Hago el producto vectorial correspondiente
            for(int k = 0; k < N; k += 1){
                C[i][j] += A[i][k]*B[k][j];
            }
        }
    }

    //Calculo el tiempo de cómputo en segundos
    double time = (double)(clock() - start) / CLOCKS_PER_SEC;
    printf("Tiempo de cómputo para N = %d: %f segundos \n", N, time);



    //Imprimo la matriz C
    // for(int i = 0;i < N; i += 1){
    //     for(int j = 0; j < N; j += 1){
    //         printf("%d ",C[i][j]);
    //     }
    //     printf("\n");
    // }


    //Libero memoria
    free(A);
    free(B);
    free(C);


    return 0;
}


